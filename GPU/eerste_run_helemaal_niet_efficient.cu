#include "hip/hip_runtime.h"

// Maximale grootte van blocks (1024)
// Ook meten van de CudaMecCpy, wat onnodig is.
// Zonder warm-up voor de meting
// Zonder average computing time



#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <chrono>


int THREADS_PER_BLOCK   1024;
int  guard_cells = 50;
int  ref_cells   = 100;
float bias       = -0.015f;      // float because data is float

// CPU
void sliding_cpu(const float* in,
                 float*       out,
                 unsigned int n,
                 int          guard_cells,
                 int          ref_cells,
                 float        bias)
{
    unsigned int limit = guard_cells + ref_cells;
    for (unsigned int i = 0; i < n; ++i) {
        if (i < limit || i >= n - limit) {
            out[i] = 0.0f;
        } else {
            float sum = 0.0f;
            for (int d = guard_cells + 1; d <= limit; ++d)
                sum += in[i - d];
            for (int d = guard_cells + 1; d <= limit; ++d)
                sum += in[i + d];
            // subtract bias and normalize
            out[i] = ((sum) / (2.0f * ref_cells)) - bias;
        }
    }
}

// GPU kernel

// ideas
// Instead of computing all the elements at once, split the array in 1024-limit blocks
// then combine them
// would this improve performance?


__global__ void sliding_gpu(const float*  in,
                            float*        out,
                            unsigned int  n,
                            int           guard_cells,
                            int           ref_cells,
                            float         bias)
{
    unsigned int idx   = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int limit = guard_cells + ref_cells;

    // If cell is under consideration (either reference or guard)
    if (idx >= limit && idx < n - limit) {
        float sum = 0.0f;  // set the sum to 0.0f (initialization)
        
        // Take the average of the reference cells
        for (int d = guard_cells + 1; d <= limit; ++d) {
            sum += in[idx - d];
            sum += in[idx + d];
        }
        out[idx] = ((sum) / (2.0f * ref_cells)) - bias;
    }
}

int main() {
    // Read data.txt → voltages vector
    std::ifstream inFile("data_clean.txt");
    if (!inFile) {
        std::cerr << "Error: could not open data.txt\n";
        return 1;
    }
    std::vector<float> voltages;
    std::string        line;
    while (std::getline(inFile, line)) {
        if (line.empty()) continue;
        std::stringstream ss(line);
        float t, v;
        char  comma;
        if (!(ss >> t >> comma >> v) || comma != ',') {
            std::cerr << "Warning: skipping malformed line: " << line << "\n";
            continue;
        }
        voltages.push_back(v);
    }
    inFile.close();

    unsigned int N = static_cast<unsigned int>(voltages.size());
    if (N == 0) {
        std::cerr << "No data loaded!\n";
        return 1;
    }

    // Create host arrays
    float *h_in      = voltages.data();          // input points at our vector
    float *h_out_cpu = new float[N];
    float *h_out_gpu = new float[N];

    // CPU timing + compute
    auto t0 = std::chrono::high_resolution_clock::now();
    sliding_cpu(h_in, h_out_cpu, N, guard_cells, ref_cells, bias);
    auto t1 = std::chrono::high_resolution_clock::now();
    double cpu_ms = std::chrono::duration<double,std::milli>(t1 - t0).count();

    // GPU setup, copy, compute
    size_t   bytes  = N * sizeof(float);
    float   *d_in, *d_out;
    hipMalloc(&d_in, bytes);
    hipMalloc(&d_out, bytes);
    hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice);
    hipMemset(d_out, 0, bytes);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    unsigned int blocks  = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    hipEventRecord(start);
    sliding_gpu<<<blocks, THREADS_PER_BLOCK>>>(d_in, d_out, N, guard_cells, ref_cells, bias);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float gpu_ms = 0.0f;
    hipEventElapsedTime(&gpu_ms, start, stop);

    hipMemcpy(h_out_gpu, d_out, bytes, hipMemcpyDeviceToHost);

    //Check & Print 
    bool match = true;
    for (unsigned int i = 0; i < N; ++i) {
        if (fabs(h_out_cpu[i] - h_out_gpu[i]) > 1e-5f) {
            match = false; break;
        }
    }

    std::cout << "CPU time: " << cpu_ms << " ms\n";
    std::cout << "GPU time: " << gpu_ms << " ms\n";
    std::cout << "Match? "    << (match ? "YES\n" : "NO\n");

    // print first few
    std::cout << "\nIndex : CPU vs GPU\n";
    for (unsigned int i = 0; i < std::min<unsigned int>(10, N); ++i) {
        std::cout << i << " : "
                  << h_out_cpu[i] << " vs "
                  << h_out_gpu[i] << "\n";
    }

    // Write to results.txt
    std::ofstream outFile("results.txt");
    if (!outFile) {
        std::cerr << "Error opening results.txt for writing\n";
    } else {
        for (unsigned int i = 0; i < N; ++i) {
            outFile << i
                    << ',' << h_out_gpu[i]
                    << '\n';
        }
        outFile.close();
        std::cout << "Results written to results.txt\n";
    }

    // Cleanup
    hipFree(d_in);
    hipFree(d_out);
    delete[] h_out_cpu;
    delete[] h_out_gpu;
    return 0;
}
